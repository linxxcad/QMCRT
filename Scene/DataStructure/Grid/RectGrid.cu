#include "hip/hip_runtime.h"
#include "RectGrid.cuh"
#include "global_function.cuh"
#include "check_cuda.h"

void RectGrid::Cinit() {
    grid_num_.x = ceil(size_.x / interval_.x);
    grid_num_.y = ceil(size_.y / interval_.y);
    grid_num_.z = ceil(size_.z / interval_.z);
}

void RectGrid::CClear() {
    if (d_grid_helio_match_) {
        checkCudaErrors(hipFree(d_grid_helio_match_));
        d_grid_helio_match_ = nullptr;
    }

    if (d_grid_helio_index_) {
        checkCudaErrors(hipFree(d_grid_helio_index_));
        d_grid_helio_index_ = nullptr;
    }
}

int boxIntersect(int mirrorBeiginId,
                 int subMirrorSize,
                 float3 min_pos,
                 float3 max_pos,
                 const RectGrid &grid,
                 vector<vector<int> > &grid_mirrow_match_vector) {
    int size = 0;
    float3 pos = grid.getPosition();
    float3 interval = grid.getInterval();
    int3 grid_num = grid.getGridNumber();

    int3 min_grid_pos = make_int3((min_pos - pos).x / interval.x,
                                  (min_pos - pos).y / interval.y,
                                  (min_pos - pos).z / interval.z);
    int3 max_grid_pos = make_int3((max_pos - pos).x / interval.x,
                                  (max_pos - pos).y / interval.y,
                                  (max_pos - pos).z / interval.z);

    if (min_grid_pos.x < 0 || min_grid_pos.y < 0 || min_grid_pos.z < 0 ||
        max_grid_pos.x >= grid_num.x || max_grid_pos.y >= grid_num.y || max_grid_pos.z >= grid_num.z) {
        std::cerr << "The heliostats may out of the grid boundary. Please check your file." << std::endl;
    }

    for (int x = max(0, min_grid_pos.x); x <= min(grid_num.x - 1, max_grid_pos.x); ++x) {
        for (int y = max(0, min_grid_pos.y); y <= min(grid_num.y - 1, max_grid_pos.y); ++y) {
            for (int z = max(0, min_grid_pos.z); z <= min(grid_num.z - 1, max_grid_pos.z); ++z) {
                int pos_id = x * grid_num.y * grid_num.z + y * grid_num.z + z;
                for (int i = 0; i < subMirrorSize; ++i) {
                    grid_mirrow_match_vector[pos_id].push_back(mirrorBeiginId + i);
                }
                size += subMirrorSize;
            }
        }
    }

    return size;
}

int RectGrid::CGridHelioMatch(
        const vector<Heliostat *> &h_helios) // set *d_grid_helio_match_, *d_grid_helio_index_ and num_grid_helio_match_
{
    if (d_grid_helio_match_ || d_grid_helio_index_) {
        throw std::runtime_error(
                "The grid and heliostats corresponding relationship should be empty before calling this method");
    }

    int start_subhelio_pos = 0;
    float3 minPos, maxPos;
    float radius = 0.0f;
    num_grid_helio_match_ = 0;

    vector<vector<int> > grid_mirrow_match_vector(grid_num_.x * grid_num_.y * grid_num_.z, vector<int>());
    for (int i = start_helio_pos_; i < start_helio_pos_ + num_helios_; ++i) {
        float3 pos = h_helios[i]->getPosition();
        radius = length(h_helios[i]->getSize()) / 2;

        minPos = pos - radius;
        maxPos = pos + radius;

        num_grid_helio_match_ += boxIntersect(start_subhelio_pos, h_helios[i]->getSubHelioSize(),
                                              minPos, maxPos, *this, grid_mirrow_match_vector);
        start_subhelio_pos += h_helios[i]->getSubHelioSize();
    }

    int *h_grid_helio_index = new int[grid_num_.x * grid_num_.y * grid_num_.z + 1];
    h_grid_helio_index[0] = 0;
    int *h_grid_helio_match = new int[num_grid_helio_match_];

    int index = 0;
    for (int i = 0; i < grid_num_.x * grid_num_.y * grid_num_.z; ++i) {
        h_grid_helio_index[i + 1] = h_grid_helio_index[i] + grid_mirrow_match_vector[i].size();
        for (int j = 0; j < grid_mirrow_match_vector[i].size(); ++j, ++index)
            h_grid_helio_match[index] = grid_mirrow_match_vector[i][j];
    }

    global_func::cpu2gpu(d_grid_helio_match_, h_grid_helio_match, num_grid_helio_match_);
    global_func::cpu2gpu(d_grid_helio_index_, h_grid_helio_index, grid_num_.x * grid_num_.y * grid_num_.z + 1);

    delete[] h_grid_helio_index;
    delete[] h_grid_helio_match;
    h_grid_helio_index = nullptr;
    h_grid_helio_match = nullptr;

    return start_subhelio_pos;
}

/**
 * Getter and setter of attributes for RectGrid
 */
void RectGrid::setGridNumber(int3 grid_num) {
    grid_num_ = grid_num;
}

void RectGrid::setDeviceGridHeliostatMatch(int *d_grid_helio_match) {
    d_grid_helio_match_ = d_grid_helio_match;
}

void RectGrid::setDeviceGridHelioIndex(int *d_grid_helio_index) {
    d_grid_helio_index_ = d_grid_helio_index;
}

size_t RectGrid::getNumberOfGridHeliostatMatch() const {
    return num_grid_helio_match_;
}

void RectGrid::setNumberOfGridHeliostatMatch(size_t num_grid_helio_match) {
    num_grid_helio_match_ = num_grid_helio_match;
}