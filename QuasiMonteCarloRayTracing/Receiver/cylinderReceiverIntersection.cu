#include "hip/hip_runtime.h"
#include "cylinderReceiverIntersection.cuh"
#include "receiverIntersectionUtil.cuh"

__device__ void cylinderReceiverIntersect::receiver_drawing(CylinderReceiver &cylinderReceiver,
        const float3 &orig, const float3 &dir, const float3 &normal, float factor) {
    //	Step1: Intersect with receiver
    float t, u, v;
    if (!cylinderReceiver.GIntersect(orig, dir, t, u, v))
        return;

    //	Step2: Calculate the energy of the light
    float energy = calEnergy(t, dir, normal, factor);

    //	Step3: Add the energy to the intersect position
    // Intersect location
    int2 row_col = make_int2(u * cylinderReceiver.getResolution().y, v * cylinderReceiver.getResolution().x);
    int address = row_col.x * cylinderReceiver.getResolution().x + row_col.y;  //col_row.y + col_row.x*resolution.y;
    float *image = cylinderReceiver.getDeviceImage();
    atomicAdd(&(image[address]), energy);
}