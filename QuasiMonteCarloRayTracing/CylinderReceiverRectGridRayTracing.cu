#include "hip/hip_runtime.h"
#include "CylinderReceiverRectGridRayTracing.cuh"
#include "Grid/rectGridDDA.cuh"
#include "Receiver/cylinderReceiverIntersection.cuh"

__global__ void map_raytracing(SunrayArgument sunrayArgument, CylinderReceiver cylinderReceiver,
                               RectGrid rectGrid, HeliostatArgument heliostatArgument,
                               float3 *d_heliostat_vertexes, float factor) {
    long long myId = global_func::getThreadId();
    if (myId >= heliostatArgument.numberOfMicroHeliostats * sunrayArgument.numberOfLightsPerGroup)
        return;

    //	Step 1: whether the incident light is shadowed by other heliostats
    int address = (heliostatArgument.d_microHelio_groups[myId / sunrayArgument.numberOfLightsPerGroup] +
                   myId % sunrayArgument.numberOfLightsPerGroup) % sunrayArgument.pool_size;
    float3 dir = global_func::local2world(sunrayArgument.d_samplelights[address], -sunrayArgument.sunray_direction);
    float3 origin = heliostatArgument.d_microHelio_origins[myId / sunrayArgument.numberOfLightsPerGroup];
    if (rectGridDDA::collision(origin, dir, rectGrid, d_heliostat_vertexes, heliostatArgument)) {
        return;
    }

    //	Step 2: whether the reflect light is shadowed by other heliostats
    float3 normal = heliostatArgument.d_microHelio_normals[myId / sunrayArgument.numberOfLightsPerGroup];
    address = (heliostatArgument.d_microHelio_groups[(myId / sunrayArgument.numberOfLightsPerGroup + 1) %
                                                     sunrayArgument.pool_size] +
               myId % sunrayArgument.numberOfLightsPerGroup) % sunrayArgument.pool_size;
    normal = global_func::local2world(sunrayArgument.d_perturbations[address], normal);
    normal = normalize(normal);

    dir = normalize(reflect(-dir, normal));
    if (rectGridDDA::collision(origin, dir, rectGrid, d_heliostat_vertexes, heliostatArgument)) {
        return;
    }

    // Step 3: intersect with receiver
    cylinderReceiverIntersect::receiver_drawing(cylinderReceiver, origin, dir, normal, factor);
}

void CylinderReceiverRectGridRayTracing(SunrayArgument &sunrayArgument, CylinderReceiver *cylinderReceiver,
                                         RectGrid *rectGrid, HeliostatArgument &heliostatArgument,
                                         float3 *d_subHeliostat_vertexes, float factor) {
    int nThreads = 512;
    dim3 nBlocks;
    global_func::setThreadsBlocks(nBlocks, nThreads,
                                  heliostatArgument.numberOfMicroHeliostats * sunrayArgument.numberOfLightsPerGroup,
                                  true);
    map_raytracing << < nBlocks, nThreads >> >
                                 (sunrayArgument, *cylinderReceiver, *rectGrid, heliostatArgument, d_subHeliostat_vertexes, factor);

    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());
}